#include "hip/hip_runtime.h"
#include "assert.h"
#include <vector>
#include <iostream>
#include "decodelayer.h"
#include "cuda_utils.h"

namespace Tn
{
    template<typename T>
    void write(char*& buffer, const T& val)
    {
        *reinterpret_cast<T*>(buffer) = val;
        buffer += sizeof(T);
    }

    template<typename T>
    void read(const char*& buffer, T& val)
    {
        val = *reinterpret_cast<const T*>(buffer);
        buffer += sizeof(T);
    }
}

namespace nvinfer1{

    DecodeLayerPlugin::DecodeLayerPlugin(int classCount, int netWidth, int netHeight, int maxOutObject, std::vector<float> anchors) {
        mClassCount = classCount;
        mYoloV7NetHeight = netHeight;
        mYoloV7NetWidth = netWidth;
        mMaxOutObject = maxOutObject;
        mAnchorLen = (int)anchors.size();
        CUDA_CHECK(hipHostMalloc(&mAnchor_h, mAnchorLen*sizeof(float)));
        CUDA_CHECK(hipMemcpy(mAnchor_h, &anchors[0], mAnchorLen*sizeof(float), hipMemcpyHostToHost));

        CUDA_CHECK(hipMalloc(&mAnchor_d, mAnchorLen*sizeof(float)));
        CUDA_CHECK(hipMemcpy(mAnchor_d, mAnchor_h, mAnchorLen*sizeof(float ), hipMemcpyHostToDevice));
    }

    DecodeLayerPlugin::~DecodeLayerPlugin() noexcept {
        CUDA_CHECK(hipHostFree(mAnchor_h));
        CUDA_CHECK(hipFree(mAnchor_d));
    }

    DecodeLayerPlugin::DecodeLayerPlugin(const void *data, size_t length) {
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mClassCount);
        read(d, mThreadCount);
        read(d, mYoloV7NetWidth);
        read(d, mYoloV7NetHeight);
        read(d, mMaxOutObject);
        read(d, mAnchorLen);
        CUDA_CHECK(hipHostMalloc(&mAnchor_h, mAnchorLen*sizeof(float)));
        memcpy(mAnchor_h, d, mAnchorLen*sizeof(float));
        d += mAnchorLen*sizeof(float);

        CUDA_CHECK(hipMalloc(&mAnchor_d, mAnchorLen*sizeof(float)));
        CUDA_CHECK(hipMemcpy(mAnchor_d, mAnchor_h, mAnchorLen*sizeof(float), hipMemcpyHostToDevice));

        assert(d == a+length);
    }

    void DecodeLayerPlugin::serialize(void* buffer) const{
        using namespace Tn;
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mClassCount);
        write(d, mThreadCount);
        write(d, mYoloV7NetWidth);
        write(d, mYoloV7NetHeight);
        write(d, mMaxOutObject);
        write(d, mAnchorLen);

        memcpy(d, mAnchor_h, mAnchorLen*sizeof(float));
        d += mAnchorLen*sizeof(float);

        assert(d == a + getSerializationSize());
    }

    size_t DecodeLayerPlugin::getSerializationSize() const {
        return sizeof(mClassCount)+sizeof(mThreadCount)+sizeof(mYoloV7NetWidth)+sizeof(mYoloV7NetHeight)+sizeof(mMaxOutObject)+sizeof(mAnchorLen)+mAnchorLen*sizeof(float);
    }

    int DecodeLayerPlugin::initialize() { return 0;}

    Dims DecodeLayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims){
        int totalSize = mMaxOutObject * 6;
        return Dims3{totalSize+1, 1, 1};
    }

    void DecodeLayerPlugin::setPluginNamespace(const char* pluginNamespace){mPluginNamespace = pluginNamespace;}

    const char* DecodeLayerPlugin::getPluginNamespace() const {return mPluginNamespace;}

    // Return the DataType of the plugin output at the requested index
    DataType DecodeLayerPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const {return DataType::kFLOAT;}

    // Return true if output tensor is broadcast across a batch.
    bool DecodeLayerPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const {return false;}

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool DecodeLayerPlugin::canBroadcastInputAcrossBatch(int inputIndex) const {return false;}

    void DecodeLayerPlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput) {}

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void DecodeLayerPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator) {}

    // Detach the plugin object from its execution context.
    void DecodeLayerPlugin::detachFromContext() {}

    const char* DecodeLayerPlugin::getPluginType() const {return "DecodeLayer_TRT";}

    const char* DecodeLayerPlugin::getPluginVersion() const {return "1";}

    void DecodeLayerPlugin::destroy() { delete this;}

    // Clone the plugin
    IPluginV2IOExt* DecodeLayerPlugin::clone() const {
        std::vector<float> anchors((float*)mAnchor_h, (float*)mAnchor_h+mAnchorLen);
        auto* p = new DecodeLayerPlugin(mClassCount, mYoloV7NetWidth, mYoloV7NetHeight, mMaxOutObject,  anchors);
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }


    __global__ void calDetection(const float *input, float* output, int noElements, const int input_w, const int input_h,
                                 int maxOutObject, const int yoloWidth, const int yoloHeight, const float* yoloAnchor,
                                 int classes, int outputElem){

        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx > noElements) return;

        int totalGrid = yoloWidth * yoloHeight;
        int bnIdx = idx / totalGrid;
        idx = idx - totalGrid*bnIdx;

        int info_len_i = 5+classes;

        const float* curInput = input + bnIdx * (info_len_i * totalGrid * 3);

        for(int k=0; k<3; k++){
            float box_prob = curInput[idx + k*info_len_i*totalGrid + 4*totalGrid];

            if(box_prob < 0.1) continue;
            int class_id = 0;
            float max_cls_prob = 0.0;

            for(int i=5; i<info_len_i; i++){
                float p = curInput[idx + k * info_len_i * totalGrid + i * totalGrid];

                if (p > max_cls_prob) {
                    max_cls_prob = p;
                    class_id = i - 5;
                }
            }

            float *res_count = output + bnIdx*outputElem;
            int count = int(atomicAdd(res_count, 1));

            if(count> maxOutObject) return;
            float *data = res_count + 1 + count*6;

            int y = idx / yoloWidth;
            int x = idx % yoloWidth;

            data[0] = (float(x) - 0.5f + 2* curInput[idx+k*info_len_i*totalGrid + 0 * totalGrid]) * float(input_w) / float(yoloWidth);
            data[1] = (float(y) - 0.5f + 2* curInput[idx+k*info_len_i*totalGrid + 1 * totalGrid]) * float(input_h) / float(yoloHeight);
            data[2] = pow(2.0f * curInput[idx + k*info_len_i * totalGrid + 2 * totalGrid], 2) * float(yoloAnchor[2*k]);
            data[3] = pow(2.0f * curInput[idx + k*info_len_i * totalGrid + 3 * totalGrid], 2) * float(yoloAnchor[2*k+1]);
            data[4] = float(class_id);
            data[5] = box_prob * max_cls_prob;
        }
    }

    void DecodeLayerPlugin::forwardGpu(const float* const* inputs, float *output, hipStream_t stream, int batchSize){
        int outputElem = 1 + mMaxOutObject * 6;
        for (int idx = 0; idx < batchSize; ++idx) {
            CUDA_CHECK(hipMemsetAsync(output + idx * outputElem, 0, sizeof(float), stream));
        }

        for(int i=0; i< 3; i++){
            int scale = 8 * int(pow(2, i));
            int yoloWidth = mYoloV7NetWidth / scale;
            int yoloHeight = mYoloV7NetHeight / scale;
            int numElem = yoloHeight * yoloWidth * batchSize;
            const float* anchor_d = (float*)mAnchor_d + 6*i;
            calDetection<<<(numElem+mThreadCount-1)/mThreadCount, mThreadCount, 0, stream>>>
            (inputs[i], output, numElem, mYoloV7NetWidth, mYoloV7NetHeight, mMaxOutObject, yoloWidth, yoloHeight, anchor_d, mClassCount, outputElem);
        }
    }


    int DecodeLayerPlugin::enqueue(int batchSize, const void* const* inputs, void** outputs, void* workspace, hipStream_t stream){
        forwardGpu((const float* const*)inputs, (float*)outputs[0], stream, batchSize);
        return 0;
    }

    PluginFieldCollection DecodePluginCreator::mFC{};
    std::vector<PluginField> DecodePluginCreator::mPluginAttributes;

    // creator
    DecodePluginCreator::DecodePluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* DecodePluginCreator::getPluginName() const{return "DecodeLayer_TRT";}

    const char* DecodePluginCreator::getPluginVersion() const {return "1"; }

    const PluginFieldCollection* DecodePluginCreator::getFieldNames() {return &mFC;}

    IPluginV2IOExt* DecodePluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc){
        assert(fc->nbFields == 2);
        assert(strcmp(fc->fields[0].name, "netInfo") == 0);
        assert(strcmp(fc->fields[1].name, "anchor") == 0);
        int *p_netInfo = (int*)(fc->fields[0].data);
        int class_count = p_netInfo[0];
        int input_w = p_netInfo[1];
        int input_h = p_netInfo[2];
        int max_output_object_count = p_netInfo[3];

        std::vector<float> anchor((float*)fc->fields[1].data, (float*)fc->fields[1].data + fc->fields[1].length);
        auto* obj = new DecodeLayerPlugin(class_count, input_w, input_h, max_output_object_count, anchor);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* DecodePluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength){
        // This object will be deleted when the network is destroyed, which will
        // call YoloLayerPlugin::destroy()
        DecodeLayerPlugin* obj = new DecodeLayerPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }
}

